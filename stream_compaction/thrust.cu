#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* device_idata = nullptr;
            int* device_odata = nullptr;
            hipMalloc((void**)&device_idata, n * sizeof(int));
            hipMalloc((void**)&device_odata, n * sizeof(int));
            hipMemset(device_odata, 0, n * sizeof(int));
            hipMemset(device_idata, 0, n * sizeof(int));
            hipMemcpy(device_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            //hipMemcpy(device_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            thrust::device_ptr<int> dev_ptr_idata(device_idata);
            thrust::device_ptr<int> dev_ptr_odata(device_odata);
            {
                SCOPED_GPU_TIMER
                thrust::exclusive_scan(dev_ptr_idata, dev_ptr_idata + n, dev_ptr_odata);
                checkCUDAError("Thrust scan failed");
            }
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

			hipMemcpy(odata, device_odata, n * sizeof(int), hipMemcpyDeviceToHost);
        }
    }
}
