#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        // TODO: __global__
        __global__ void kernNaiveScan(int n, int pow2dminusone, int *odata, const int *idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= pow2dminusone) {
                int temp = idata[index];
                int temp1 = idata[index - pow2dminusone];
                odata[index] = temp + temp1;
            }
		}

        __global__ void kernToExclusive(int n, int* odata, const int* idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) return;
            odata[index] -= idata[index];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			const int blockSize = BLOCK_SIZE;
            int padSize = (int)pow(2.f, ilog2ceil(n));
			dim3 gridDim((padSize + blockSize - 1) / blockSize);
			dim3 blockDim(blockSize);
			//std::cout << "padsize " << padSize << std::endl;
			int* device_idata = nullptr;
			int* device_odata = nullptr;
			hipMalloc((void**)&device_idata, padSize * sizeof(int));
			hipMalloc((void**)&device_odata, padSize * sizeof(int));
			hipMemset(device_odata, 0, padSize * sizeof(int));
            hipMemset(device_idata, 0, padSize * sizeof(int));
            hipMemcpy(device_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(device_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("naive init failed!");
			{
				SCOPED_GPU_TIMER
                for (int pow2dminusone = 1; pow2dminusone < padSize; pow2dminusone <<= 1)
                {
                    kernNaiveScan << <gridDim, blockDim >> > (n, pow2dminusone, device_odata, device_idata);
                    hipMemcpy(device_idata, device_odata, padSize * sizeof(int), hipMemcpyHostToDevice);
                    checkCUDAError("kernNaiveScan init failed!");
                }
                hipMemcpy(device_odata+1, device_idata, (n-1) * sizeof(int), hipMemcpyHostToDevice);
                hipMemset(device_odata, 0, sizeof(int));
			}
            hipMemcpy(odata, device_odata, n * sizeof(int), hipMemcpyDeviceToHost);

            checkCUDAError("hipMemcpy init failed!");
        }
    }
}
